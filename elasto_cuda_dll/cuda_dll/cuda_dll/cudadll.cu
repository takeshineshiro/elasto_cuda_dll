#include "stdafx.h" //����Ԥ����ͷ�ļ�

#include "CUDAdll.cuh" //���뵼����������ͷ�ļ�



//��ʼ��CUDA
extern int count = 0;


bool InitCUDA(void)//CUDA��ʼ������
{

	printf("Start to detecte devices.........\n");//��ʾ��⵽���豸��

	hipGetDeviceCount(&count);//�������������ڵ���1.0���豸��

	if (count == 0){

		fprintf(stderr, "There is no device.\n");

		return false;

	}


	printf("%d device/s detected.\n", count);//��ʾ��⵽���豸��


	int i;

	for (i = 0; i < count; i++){//������֤��⵽���豸�Ƿ�֧��CUDA

		hipDeviceProp_t prop;

		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {//����豸���Բ���֤�Ƿ���ȷ

			if (prop.major >= 1)//��֤�����������������������ĵ�һλ���Ƿ����1

			{
				printf("Device %d: %s supports CUDA %d.%d.\n", i + 1, prop.name, prop.major, prop.minor);//��ʾ��⵽���豸֧�ֵ�CUDA�汾
				break;


			}
		}
	}

	if (i == count) {//û��֧��CUDA1.x���豸
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);//�����豸Ϊ�����̵߳ĵ�ǰ�豸
	return true;
}


void showHelloCuda(void)//����CUDA��ʼ������
{
	if (!InitCUDA()) //��ʼ��ʧ��
	{
		printf("Sorry,CUDA has not been initialized.\n");
		return;
	}
	printf("Hello GPU! CUDA has been initialized.\n");
}